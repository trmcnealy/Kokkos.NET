#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "MemoryMapped.h"

#define ROWNUM 11997996
#define REDUCTION_BLOCK_SIZE 512
#define SEGSIZE ROWNUM / 2

// Struct for holding dates when reading table
// attributes from the Q6 benchmark
struct entry
{
    char words[11];
};

/*

   Read dates from file using MemoryMapping library.

*/
void mmread_date(const std::string& filename, struct entry* date_h)
{
    MemoryMapped data(filename);
    if(!data.IsValid())
    {
        printf("Failed to read file.\n");
        return;
    }
    char* buffer = (char*)data.GetData();

    int index = 0;
    for(uint64_t i = 0; i < data.Size(); i += 11)
    { // length of date(10) + '\n'(1) = 11
        for(int j = 0; j < 10; j++)
            date_h[index].words[j] = buffer[i + j];
        date_h[index].words[10] = '\0';
        index++;
    }
    return;
}

/*
   Read floats from file using MemoryMapping library.
*/
void mmread_float(const std::string& filename, float* dest)
{
    MemoryMapped data(filename);
    if(!data.IsValid())
    {
        printf("Failed to read file.\n");
        return;
    }
    char* buffer = (char*)data.GetData();

    int  index    = 0;
    int  start    = 0;
    char temp[15] = {};
    for(uint64_t i = 0; i < data.Size(); i++)
    {
        if(buffer[i] == '\n')
        {
            int length = i - start;
            for(int j = 0; j < length; j++)
                temp[j] = buffer[start + j];
            temp[length] = '\0';
            dest[index]  = atof(temp);
            start        = i + 1;
            index++;
        }
    }
    return;
}

/*
   Query_Scan --
      This kernel scans disc_d, qty_d, price_d, and date_d
      according to the mySQL Q6 query from the TPC-H specification, described below:

   select sum(l_extendedprice*l_discount)
   as revenue from lineitem
   where(l_shipdate >= '1994-01-01' and
         l_shipdate < '1995-01-01'  and
         l_discount >= 0.05         and
         l_discount <= 0.0750       and
         l_quantity < 24;
*/
__global__ void query_scan(float* disc_d, float* qty_d, float* price_d, struct entry* date_d, float* revenue_d)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < SEGSIZE)
    {
        int count = 0;
        if(date_d[index].words[3] == '4')
        {
            count++;
        }

        if((disc_d[index] >= 0.0500) && (disc_d[index] <= 0.0750))
        {
            count++;
        }

        if(qty_d[index] < 24)
        {
            count++;
        }

        if(count == 3)
        {
            revenue_d[index] = price_d[index] * disc_d[index];
        }
        else
            revenue_d[index] = 0;
    }

    __syncthreads();
}

/*

   Query_Reduction --
      Performs a reduction sum on revenue_d, storing the respective sum
         for the current block at block_sum_d[i]
*/
__global__ void query_reduction(float* revenue_d, float* block_sum_d)
{
    __shared__ float partialSum[2 * REDUCTION_BLOCK_SIZE];

    int t     = threadIdx.x;
    int start = 2 * blockDim.x * blockIdx.x;

    if(start + t < SEGSIZE)
    {
        partialSum[t] = revenue_d[start + t];
    }
    else
    {
        partialSum[t] = 0.0;
    }

    if(start + t + REDUCTION_BLOCK_SIZE < SEGSIZE)
    {
        partialSum[t + blockDim.x] = revenue_d[start + t + blockDim.x];
    }
    else
    {
        partialSum[t + blockDim.x] = 0.0;
    }

    int stride;
    for(stride = blockDim.x; stride >= 1; stride >>= 1)
    {
        __syncthreads();

        if(t < stride)
        {
            partialSum[t] += partialSum[t + stride];
        }
    }

    if(stride == 0)
    {
        block_sum_d[blockIdx.x] = partialSum[0];
    }
}

/*

   main() --
      Main entry point to the program.
*/
int main()
{
    // Declare pointers for TPC-H benchmark data to be imported.
    // This is for discount, quantity, price, and date attributes
    //    for the Q6 query.
    float*        disc_h; // = (float*)malloc(ROWNUM*sizeof(float));
    float*        qty_h; // = (float*)malloc(ROWNUM*sizeof(float));
    float*        price_h; //= (float*)malloc(ROWNUM*sizeof(float));
    struct entry* date_h; // =(struct entry*) malloc(ROWNUM*sizeof(struct entry));

    // We're using pinned memory
    hipHostAlloc((void**)&disc_h, ROWNUM * sizeof(float), 0);
    hipHostAlloc((void**)&qty_h, ROWNUM * sizeof(float), 0);
    hipHostAlloc((void**)&price_h, ROWNUM * sizeof(float), 0);
    hipHostAlloc((void**)&date_h, ROWNUM * sizeof(struct entry), 0);

    // Define variables for holding query results.
    int   i;
    float total_revenue = 0.0;

    // define variable for stream 0
    float *       disc_d0, *qty_d0, *price_d0, *revenue_d0;
    struct entry* date_d0;

    // define variable for stream 1
    float *       disc_d1, *qty_d1, *price_d1, *revenue_d1;
    struct entry* date_d1;

    dim3 dim_grid, dim_block;

    // allocate device memory for stream 0
    hipMalloc((void**)&disc_d0, SEGSIZE * sizeof(float));
    hipMalloc((void**)&qty_d0, SEGSIZE * sizeof(float));
    hipMalloc((void**)&price_d0, SEGSIZE * sizeof(float));
    hipMalloc((void**)&date_d0, SEGSIZE * sizeof(struct entry));
    hipMalloc((void**)&revenue_d0, SEGSIZE * sizeof(float));

    // allocate device memory for stream 1
    hipMalloc((void**)&disc_d1, SEGSIZE * sizeof(float));
    hipMalloc((void**)&qty_d1, SEGSIZE * sizeof(float));
    hipMalloc((void**)&price_d1, SEGSIZE * sizeof(float));
    hipMalloc((void**)&date_d1, SEGSIZE * sizeof(struct entry));
    hipMalloc((void**)&revenue_d1, SEGSIZE * sizeof(float));

    // measure the total response time
    float       responseTime;
    hipEvent_t start0, stop0;
    hipEventCreate(&start0);
    hipEventCreate(&stop0);
    hipEventRecord(start0, 0);

    // data transfer from disk to host memory using memory mapping library
    mmread_float("L_DISCOUNT.txt", disc_h);
    mmread_float("L_QUANTITY.txt", qty_h);
    mmread_float("L_EXTENDEDPRICE.txt", price_h);
    mmread_date("L_SHIPDATE.txt", date_h);

    // define multi-streams
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // decide query_scan kernel function's dimension
    dim_block.x = 1024;
    dim_block.y = dim_block.z = 1;

    dim_grid.x = SEGSIZE / 1024;
    if(SEGSIZE % 1024 != 0)
    {
        dim_grid.x++;
    }
    dim_grid.y = dim_grid.z = 1;

    // decide query_reduction kernel function's dimension
    int    num_blocks = ceil((float)SEGSIZE / (REDUCTION_BLOCK_SIZE * 2));
    float *block_sum_d0, *block_sum_d1;
    float* block_sum_h;

    // allocate memory for block_sum_h
    block_sum_h = (float*)malloc(2 * num_blocks * sizeof(float));

    // allocate meory for block_sum_d0 and block_sum_d1
    hipMalloc((void**)&block_sum_d0, num_blocks * sizeof(float));
    hipMalloc((void**)&block_sum_d1, num_blocks * sizeof(float));

    // call kernel function
    // Record GPU Time.
    float       gpuTimeDMATime;
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2, 0);

    // call kenerl query_scan function to scan the whole database table
    // fetch the required data tuples

    // copy data from host to device for stream 0
    hipMemcpyAsync(disc_d0, disc_h, SEGSIZE * sizeof(float), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(qty_d0, qty_h, SEGSIZE * sizeof(float), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(price_d0, price_h, SEGSIZE * sizeof(float), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(date_d0, date_h, SEGSIZE * sizeof(struct entry), hipMemcpyHostToDevice, stream0);

    // copy data from host to device for stream 1
    hipMemcpyAsync(disc_d1, disc_h + SEGSIZE, SEGSIZE * sizeof(float), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(qty_d1, qty_h + SEGSIZE, SEGSIZE * sizeof(float), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(price_d1, price_h + SEGSIZE, SEGSIZE * sizeof(float), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(date_d1, date_h + SEGSIZE, SEGSIZE * sizeof(struct entry), hipMemcpyHostToDevice, stream1);

    // query_scan<<<dim_grid, dim_block>>>(disc_d, qty_d, price_d, date_d, revenue_d);
    query_scan<<<dim_grid, dim_block, 0, stream0>>>(disc_d0, qty_d0, price_d0, date_d0, revenue_d0);
    query_scan<<<dim_grid, dim_block, 0, stream1>>>(disc_d1, qty_d1, price_d1, date_d1, revenue_d1);

    // Synchronize between kernel calls.
    hipDeviceSynchronize();

    query_reduction<<<num_blocks, REDUCTION_BLOCK_SIZE, 0, stream0>>>(revenue_d0, block_sum_d0);
    query_reduction<<<num_blocks, REDUCTION_BLOCK_SIZE, 0, stream1>>>(revenue_d1, block_sum_d1);

    // Mark GPU end time
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&gpuTimeDMATime, start2, stop2);

    // Copy results back to host, calculate total revenue.
    hipMemcpyAsync(block_sum_h, block_sum_d0, num_blocks * sizeof(float), hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(block_sum_h + num_blocks, block_sum_d1, num_blocks * sizeof(float), hipMemcpyDeviceToHost, stream1);

    // measure execution time, GPU time, and CPU time
    float       cpuTime;
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1, 0);

    for(i = 0; i < 2 * num_blocks; i++)
    {
        total_revenue = total_revenue + block_sum_h[i];
    }

    // Mark execution end time
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&cpuTime, start1, stop1);

    // Output query result
    printf("\n");
    printf("+----------+\n");
    printf("| revenue  |\n");
    printf("+----------+\n");
    printf(" %f\n", total_revenue);
    printf("+----------+\n");

    // Free all allocated resources
    hipHostFree(disc_h);
    hipHostFree(qty_h);
    hipHostFree(price_h);
    hipHostFree(date_h);

    free(block_sum_h);

    hipFree(disc_d0);
    hipFree(qty_d0);
    hipFree(price_d0);
    hipFree(date_d0);
    hipFree(revenue_d0);
    hipFree(block_sum_d0);

    hipFree(disc_d1);
    hipFree(qty_d1);
    hipFree(price_d1);
    hipFree(date_d1);
    hipFree(revenue_d1);
    hipFree(block_sum_d1);

    // Stop timer for response time
    hipEventRecord(stop0, 0);
    hipEventSynchronize(stop0);
    hipEventElapsedTime(&responseTime, start0, stop0);

    printf("the total response time is: %f ms\n", responseTime);
    printf("the gpu + DMA time is: %f ms\n", gpuTimeDMATime);
    printf("the cpu time is: %f ms\n", cpuTime);
    // printf("the DMA time is: %f ms\n", DMATime);
    printf("the IO time is: %f ms\n", responseTime - gpuTimeDMATime - cpuTime);
    return 0;
}
