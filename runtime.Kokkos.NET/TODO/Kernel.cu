#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 *
 * Licensed under the MIT License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <typeinfo>
#include <hip/hip_fp16.h>

#include "inc/Core/Common/cuda/params.h"
#include "inc/Core/Common/cuda/TPtree.hxx"

/*****************************************************************************************
* Count the number of points assigned to each leaf
*****************************************************************************************/
__global__ void count_leaf_sizes(LeafNode* leafs, int* node_ids, int N, int internal_nodes) {
    int leaf_id;
    for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < N; i += blockDim.x*gridDim.x) {
        leaf_id = node_ids[i] - internal_nodes;
        atomicAdd(&leafs[leaf_id].size, 1);
    }
}

/*****************************************************************************************
* Assign each point to a leaf node (based on its node_id when creating the tptree).  Also
* computes the size and offset of each leaf node for easy permutation.
*****************************************************************************************/
__global__ void assign_leaf_points(LeafNode* leafs, int* leaf_points, int* node_ids, int N, int internal_nodes) {
    int leaf_id;
    int idx;
    for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < N; i += blockDim.x*gridDim.x) {
        leaf_id = node_ids[i] - internal_nodes;
        idx = atomicAdd(&leafs[leaf_id].size, 1);
        leaf_points[idx + leafs[leaf_id].offset] = i;
    }
}


__global__ void assign_leaf_points_in_batch(LeafNode* leafs, int* leaf_points, int* node_ids, int N, int internal_nodes, int min_id, int max_id) {
    int leaf_id;
    int idx;
    for (int i = min_id + blockIdx.x*blockDim.x + threadIdx.x; i < max_id; i += blockDim.x*gridDim.x) {
        leaf_id = node_ids[i] - internal_nodes;
        idx = atomicAdd(&leafs[leaf_id].size, 1);
        leaf_points[idx + leafs[leaf_id].offset] = i;
    }
}

__global__ void assign_leaf_points_out_batch(LeafNode* leafs, int* leaf_points, int* node_ids, int N, int internal_nodes, int min_id, int max_id) {
    int leaf_id;
    int idx;
    for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < min_id; i += blockDim.x*gridDim.x) {
        leaf_id = node_ids[i] - internal_nodes;
        idx = atomicAdd(&leafs[leaf_id].size, 1);
        leaf_points[idx + leafs[leaf_id].offset] = i;
    }

    for (int i = max_id + blockIdx.x*blockDim.x + threadIdx.x; i < N; i += blockDim.x*gridDim.x) {
        leaf_id = node_ids[i] - internal_nodes;
        idx = atomicAdd(&leafs[leaf_id].size, 1);
        leaf_points[idx + leafs[leaf_id].offset] = i;
    }
}

